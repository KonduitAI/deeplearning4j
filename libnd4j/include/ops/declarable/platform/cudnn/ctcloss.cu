/*******************************************************************************
 *
 * Copyright (c) 2021 Konduit K.K.
 *
 * This program and the accompanying materials are made available under the
 * terms of the Apache License, Version 2.0 which is available at
 * https://www.apache.org/licenses/LICENSE-2.0.
 *
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS, WITHOUT
 * WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied. See the
 * License for the specific language governing permissions and limitations
 * under the License.
 *
 * SPDX-License-Identifier: Apache-2.0
 ******************************************************************************/
 //
 // @author AbdelRauf
 //

#include "cudnnUtils.h"
#include <vector>


namespace sd   {
namespace ops     {
namespace platforms {



    template<typename Op, typename ...Args>
    void callCudnnIfNoErr(hipdnnStatus_t &err, Op op, Args&&... args){
        if(err==HIPDNN_STATUS_SUCCESS){
            err = op(std::forward<Args>(args)...);
            if(err){
                nd4j_printf("Cudnn error code %s\n",hipdnnGetErrorString(err));
            }
        }
    }

    template <typename T>
    const T* bufferInHost( const NDArray *array)  {
        array->syncToHost();
        return reinterpret_cast<const T*>(array->buffer());
    }

    std::vector<int> getConcatTargets(const NDArray &targetLabels, const NDArray &targetLabelLengths){
                //concatenate target labels
                const int32_t *tlabels = bufferInHost<int32_t>(&targetLabels);
                const int32_t *tlens =bufferInHost<int32_t>(&targetLabelLengths);
                int32_t nextOffset = targetLabels.strideAt(0);
                int32_t elStride = targetLabels.strideAt(1);
                int32_t batchCount = targetLabelLengths.lengthOf();
                std::vector<int> labels;
                labels.resize(targetLabels.lengthOf());
                int j=0;
                if(targetLabels.ews()){
                    for(int i=0; i<batchCount;i++){
                        int count = tlens[i];
                        for( int k=0;k<count;k++){
                            labels[j] = tlabels[k];
                            j++;
                        }
                        tlabels+=nextOffset;
                    }
                }else{
                    for(int i=0; i<batchCount;i++){
                        int count = tlens[i];
                        for( int k=0;k<count;k++){
                            labels[j] = tlabels[k*elStride];
                            j++;
                        }
                        tlabels+=nextOffset;
                    }
                }
                return labels;
    }


    PLATFORM_IMPL(ctc_loss, ENGINE_CUDA) {
        auto targetLabels = INPUT_VARIABLE(0);
        auto logitInput = INPUT_VARIABLE(1);
        auto targetLabelLengths = INPUT_VARIABLE(2);
        auto logitInputLengths = INPUT_VARIABLE(3); 
        auto outputLosses = OUTPUT_VARIABLE(0);
        int blankIndex = INT_ARG(0);
        auto context = block.launchContext();
        auto handle = reinterpret_cast<hipdnnHandle_t *>(context->getCuDnnHandle());

        hipdnnStatus_t err = HIPDNN_STATUS_SUCCESS;
        hipdnnSetStream(*handle, *context->getCudaStream());

        //in Cudnn inputs are probabilities
        //in Cudnn Batch is in the middle dimension
        auto probs = logitInput->ulike();
        logitInput->applyTransform(sd::transform::Exp, probs);
        probs.permutei({1,0,2}); 
        const int dims[] = {(int)probs.sizeAt(0), (int)probs.sizeAt(1), (int)probs.sizeAt(2)};
        const int strides[] = {(int)probs.strideAt(0), (int)probs.strideAt(1), (int)probs.strideAt(2)};

        //in Cudnn targets are concantenated instead of batched as matrix
        auto labels = getConcatTargets(*targetLabels, *targetLabelLengths);
        const int32_t * ldata= labels.data();

        cudnnCTCLossDescriptor_t  ctcLossDesc;
        hipdnnTensorDescriptor_t probsDesc;
        callCudnnIfNoErr(err,cudnnCreateCTCLossDescriptor,&ctcLossDesc);
        callCudnnIfNoErr(err,hipdnnCreateTensorDescriptor,&probsDesc);
        callCudnnIfNoErr(err, hipdnnSetTensorNdDescriptor,probsDesc, cudnnDataType(logitInput->dataType()), logitInput->rankOf() , dims, strides);


        size_t tempWorkSpaceSize=0;
        callCudnnIfNoErr(err,cudnnGetCTCLossWorkspaceSize, *handle,  probsDesc, nullptr,
            ldata,
            bufferInHost<int32_t>(targetLabelLengths),
            bufferInHost<int32_t>(logitInputLengths),
            CUDNN_CTC_LOSS_ALGO_DETERMINISTIC,
            ctcLossDesc, &tempWorkSpaceSize);

        // Allocate temp tempWorkspace buffer
        void *tempWorkSpace = nullptr;
        hipMalloc(&tempWorkSpace, tempWorkSpaceSize);

        NDArray::prepareSpecialUse({outputLosses}, {logitInput});
        callCudnnIfNoErr(err, cudnnCTCLoss,*handle,
            probsDesc,
            probs.specialBuffer(),
            ldata,
            bufferInHost<int32_t>(targetLabelLengths),
            bufferInHost<int32_t>(logitInputLengths),
            outputLosses->specialBuffer(),
            nullptr,
            nullptr,
            CUDNN_CTC_LOSS_ALGO_DETERMINISTIC,
            ctcLossDesc,
            tempWorkSpace,
            tempWorkSpaceSize);

        NDArray::registerSpecialUse({outputLosses}, {logitInput});

        hipFree(tempWorkSpace);
        callCudnnIfNoErr(err, hipdnnDestroyTensorDescriptor,probsDesc);
        callCudnnIfNoErr(err, cudnnDestroyCTCLossDescriptor,ctcLossDesc);

        if(err!=HIPDNN_STATUS_SUCCESS) throw sd::cuda_exception::build("ctc_loss CUDNN call failure ", err);
        return Status::OK();
    }

    template<typename T>
    bool checkLabelLength(const NDArray &labelLengthArr){
            //check label lengthes
            auto lenBatch = labelLengthArr.lengthOf(); 
            for(int i=0; i < lenBatch; i++){
                // The labelLengths is greater than 256.
                if(labelLengthArr.e<int32_t>(i)>256) return false;
            }
            return true;
    }

    PLATFORM_CHECK(ctc_loss, ENGINE_CUDA) {
        auto targetLabels = INPUT_VARIABLE(0);
        auto logitInput = INPUT_VARIABLE(1);
        auto targetLabelLengths = INPUT_VARIABLE(2);
        auto logitInputLengths = INPUT_VARIABLE(3); 
        auto outputLosses = OUTPUT_VARIABLE(0);
        int blankIndex = INT_ARG(0);

        auto dTypeInput = logitInput->dataType();
        auto intType = targetLabelLengths->dataType();
        auto dTypeOutput = outputLosses->dataType();

        bool is_supported = blankIndex==0 && intType == DataType::INT32  && dTypeInput == DataType::FLOAT32;
        is_supported = is_supported && outputLosses->ews() && targetLabelLengths->ews() && targetLabels->ews() && logitInputLengths->ews();
        if(is_supported){
            is_supported = is_supported && checkLabelLength<int32_t>(*targetLabelLengths);
        }
        return  is_supported; 
    }

    PLATFORM_IMPL(ctc_loss_grad, ENGINE_CUDA) {
        return Status::OK();
    }

    PLATFORM_CHECK(ctc_loss_grad, ENGINE_CUDA) {
        return false;
    } 

}
}
}