#include "hip/hip_runtime.h"
/*******************************************************************************
 * Copyright (c) 2019 Konduit K.K.
 *
 * This program and the accompanying materials are made available under the
 * terms of the Apache License, Version 2.0 which is available at
 * https://www.apache.org/licenses/LICENSE-2.0.
 *
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS, WITHOUT
 * WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied. See the
 * License for the specific language governing permissions and limitations
 * under the License.
 *
 * SPDX-License-Identifier: Apache-2.0
 ******************************************************************************/

//
// @author Yurii Shyrma (iuriish@yahoo.com)
//

#include "cudnnUtils.h"
#include<ops/declarable/helpers/transforms.h>

namespace nd4j      {
namespace ops       {
namespace platforms {

//////////////////////////////////////////////////////////////////////////
static void lstmLayerCUDNN(const LaunchContext* context,
                        const NDArray* x, const NDArray* w, const NDArray* b, const NDArray* seqLen, const NDArray* hI, const NDArray* cI,
                        NDArray* h, NDArray* hL, NDArray* cL,
                        const std::vector<float>& params) {
    // notations:
    // bS - batch size
    // sL - sequence length, number of time steps
    // nIn - input size
    // nOut - output size (hidden size)

    //     INPUTS:

    // *******
    // input x:
    // 1) [bS, nIn, sL]  when dataFormat == 2

    // *******
    // weights (input + recurrent) w:
    // 1) [nIn + nOut, 4*nOut]    when directionMode <  2
    // 2) [2*(nIn + nOut), 4*nOut] when directionMode >= 2

    // *******
    // biases b:
    // 1) [4*nOut]    when directionMode <  2
    // 2) [2, 4*nOut] when directionMode >= 2

    // *******
    // sequence length array seqLen:
    // 1) [bS] always

    // *******
    // initial output hI:
    // 1) [bS, nOut]    when directionMode <  2
    // 2) [2, bS, nOut] when directionMode >= 2

    // *******
    // initial cell state cI (same shape as in hI):
    // 1) [bS, nOut]    when directionMode <  2
    // 2) [2, bS, nOut] when directionMode >= 2


    //     OUTPUTS:

    // *******
    // output h:
    // 1) [bS, nOut, sL]    when directionMode <= 2 && dataFormat == 2
    // 2) [bS, 2*nOut, sL]  when directionMode == 3 && dataFormat == 2

    // *******
    // output at last step hL:
    // 1) [bS, nOut]    when directionMode <  2
    // 2) [2, bS, nOut] when directionMode >= 2

    // *******
    // cell state at last step cL (same shape as in hL):
    // 1) [bS, nOut]    when directionMode <  2
    // 2) [2, bS, nOut] when directionMode >= 2

    // !!! dimension 4*nOut implies order it, ft, c't, ot
    // !!! dimension 3*nOut implies order it, ft, ot

    auto handle = reinterpret_cast<hipdnnHandle_t *>(context->getCuDnnHandle());
    hipdnnStatus_t err = hipdnnSetStream(*handle, *context->getCudaStream());
    if (err != 0) throw nd4j::cuda_exception::build("conv2dCUDNN: can't set stream for cuDNN", err);

    const int numDims = 3;

    const int numOfDirections = (params[1] /*directionMode*/ == 0 ) ? 1 : 2;

    const int bS   = x->sizeAt(0);
    const int nIn  = x->sizeAt(1);
    const int sL   = x->sizeAt(2);
    const int nOut = w->sizeAt(-1) / 4;

    int* seqLengthArray = nullptr;
    if(seqLen != nullptr)
        seqLengthArray = seqLen->bufferAsT<int>();
    else {
        seqLengthArray = new int[bS];
        for (uint i = 0; i < bS; ++i)
            seqLengthArray[i] = sL;
    }

    // const std::vector<int> xShape = {bS, nIn, sL};
    // const std::vector<int> hShape = {bS, numOfDirections*nOut, sL};
    const std::vector<int> wShape = {1, numOfDirections*(nIn + nOut), 4*nOut};
    const std::vector<int> hIcIShape = {numOfDirections, bS, nOut};

    hipdnnTensorFormat_t format = HIPDNN_TENSOR_NCHW;

    // x descriptor
    cudnnRNNDataDescriptor_t xDesc;
    cudnnCreateRNNDataDescriptor(&xDesc);
    err = cudnnSetRNNDataDescriptor(xDesc, cudnnDataType(x->dataType()), CUDNN_RNN_DATA_LAYOUT_SEQ_MAJOR_PACKED, sL, bS, nIn, seqLengthArray, nullptr);
    if(err != 0) throw nd4j::cuda_exception::build("lstmLayerCUDNN: cudnnSetRNNDataDescriptor for x failed", err);

    // hI/cI/hL/cL, descriptor is same for all of them
    hipdnnTensorDescriptor_t hcDesc;
    if(hI != nullptr || cI != nullptr || cL != nullptr || cL != nullptr) {
        hipdnnCreateTensorDescriptor(&hcDesc);
        err = cudnnSetTensorNdDescriptorEx(hcDesc, format, cudnnDataType(hI != nullptr ? hI->dataType() : cI->dataType()), numDims, hIcIShape.data());
        if(err != 0) throw nd4j::cuda_exception::build("lstmLayerCUDNN: hipdnnSetFilterNdDescriptor for hI/cI/hL/cL failed", err);
    }

    // weights descriptor
    hipdnnFilterDescriptor_t wDesc;
    hipdnnCreateFilterDescriptor(&wDesc);
    err = hipdnnSetFilterNdDescriptor(wDesc, cudnnDataType(w->dataType()), format, numDims, wShape.data());
    if(err != 0) throw nd4j::cuda_exception::build("lstmLayerCUDNN: hipdnnSetFilterNdDescriptor for weights failed", err);

    // h descriptor
    cudnnRNNDataDescriptor_t hDesc;
    cudnnCreateRNNDataDescriptor(&hDesc);
    err = cudnnSetRNNDataDescriptor(hDesc, cudnnDataType(h->dataType()), CUDNN_RNN_DATA_LAYOUT_SEQ_MAJOR_PACKED, sL, bS, nOut, seqLengthArray, nullptr);
    if(err != 0) throw nd4j::cuda_exception::build("lstmLayerCUDNN: cudnnSetRNNDataDescriptor for h failed", err);

    // no dropout descriptor
    hipdnnDropoutDescriptor_t dropoutDesc;

    // description of lstm
    hipdnnDataType_t typeOfData = HIPDNN_DATA_FLOAT;
    if(x->dataType() == DataType::DOUBLE)
        typeOfData = HIPDNN_DATA_DOUBLE;
    else if(x->dataType() == DataType::HALF)
        typeOfData = HIPDNN_DATA_HALF;

    hipdnnRNNDescriptor_t lstmDesc;
    hipdnnCreateRNNDescriptor(&lstmDesc);
    err = hipdnnSetRNNDescriptor(*handle, lstmDesc, nOut, 1, dropoutDesc, HIPDNN_LINEAR_INPUT,
                                (numOfDirections == 1) ? HIPDNN_UNIDIRECTIONAL : HIPDNN_BIDIRECTIONAL,
                                HIPDNN_LSTM, HIPDNN_RNN_ALGO_STANDARD, typeOfData);
    if (err != 0) throw nd4j::cuda_exception::build("lstmLayerCUDNN: hipdnnSetRNNDescriptor failed", err);

    // set clip value
    if(params[2] != 0) {    // params[2] == clipValue
        err = cudnnRNNSetClip(*handle, lstmDesc, CUDNN_RNN_CLIP_MINMAX, HIPDNN_NOT_PROPAGATE_NAN, -params[2], params[2]);
        if (err != 0) throw nd4j::cuda_exception::build("lstmLayerCUDNN: cudnnRNNSetClip failed", err);
    }

    // set bias mode
    // if(b != nullptr) {
    //     err = hipdnnStatus_t cudnnSetRNNBiasMode(lstmDesc, HIPDNN_RNN_WITH_BIAS);
    //     if (err != 0) throw nd4j::cuda_exception::build("lstmLayerCUDNN: cudnnSetRNNBiasMode failed", err);
    // }

    // allocate amount of device memory necessary for lstm calculation process
    size_t workSpaceSizeInBytes;
    workSpaceSizeInBytes = x->lengthOf() * x->sizeOfT();
    // err = hipdnnGetRNNWorkspaceSize(*handle, lstmDesc, sL, xDesc, &workSpaceSizeInBytes);
    // if (err != 0) throw nd4j::cuda_exception::build("lstmLayerCUDNN: hipdnnGetRNNWorkspaceSize failed", err);
    void* workSpace;
    auto cudaErr = hipMalloc(&workSpace, workSpaceSizeInBytes);
    if (cudaErr != 0) throw nd4j::cuda_exception::build("lstmLayerCUDNN: hipMalloc for auxiliary workspace memory failed", cudaErr);


    NDArray::prepareSpecialUse({h, hL, cL}, {x, w, b, seqLen, hI, cI});

    // run calculation
    err = cudnnRNNForwardInferenceEx(*handle, lstmDesc,
                                     xDesc, x->getSpecialBuffer(),
                                     hcDesc, hI ? hI->getSpecialBuffer() : nullptr,
                                     hcDesc, cI ? cI->getSpecialBuffer() : nullptr,
                                     wDesc, w->getSpecialBuffer(),
                                     hDesc, h->getSpecialBuffer(),
                                     hcDesc, hL ? hL->getSpecialBuffer() : nullptr,
                                     hcDesc, cL ? cL->getSpecialBuffer() : nullptr,
                                     nullptr, nullptr, nullptr, nullptr, nullptr, nullptr, nullptr, nullptr,
                                     workSpace, workSpaceSizeInBytes);
    if (err != 0) throw nd4j::cuda_exception::build("lstmLayerCUDNN: cudnnRNNForwardInferenceEx failed", err);

    cudaErr = hipStreamSynchronize(*context->getCudaStream());
    if (cudaErr != 0)
        throw cuda_exception::build("lstmLayerCUDNN: hipStreamSynchronize failed !", cudaErr);

    NDArray::registerSpecialUse({h, hL, cL}, {x, w, b, seqLen, hI, cI});

    if(seqLen = nullptr)
        delete []seqLengthArray;
}

//////////////////////////////////////////////////////////////////////////
PLATFORM_IMPL(lstmLayer, ENGINE_CUDA) {

   // equations (no peephole connections)
    // it  = σ(Wxi * xt  +  Wri * ht-1  +  bi)
    // ft  = σ(Wxf * xt  +  Wrf * ht-1  +  bf)
    // c't = tanh(Wxc * xt  +  Wrc * ht-1  +  bc)
    // ct  = ft ◦ ct-1 + it ◦ c't
    // ot  = σ(Wxo * xt  +  Wro * ht-1  +  bo)
    // ht  = ot ◦ tanh(ct)

    // equations (peephole connections are present)
    // it  = σ(Wxi * xt  +  Wri * ht-1  +  Wpi ◦ ct-1  +  bi)
    // ft  = σ(Wxf * xt  +  Wrf * ht-1  +  Wpf ◦ ct-1  +  bf)
    // c't = tanh(Wxc * xt  +  Wrc * ht-1  +  bc)
    // ct  = ft ◦ ct-1 + it ◦ c't
    // ot  = σ(Wxo * xt  +  Wro * ht-1  +  Wpo ◦ ct  +  bo)
    // ht  = ot ◦ tanh(ct)

    // notations:
    // bS - batch size
    // sL - sequence length, number of time steps
    // nIn - input size
    // nOut - output size (hidden size)

    //     INPUTS:

    // *******
    // input x:
    // 1) [sL, bS, nIn]  when dataFormat == 0
    // 2) [bS, sL, nIn]  when dataFormat == 1
    // 3) [bS, nIn, sL]  when dataFormat == 2

    // *******
    // input weights Wx:
    // 1) [nIn, 4*nOut]    when directionMode <  2
    // 2) [2, nIn, 4*nOut] when directionMode >= 2

    // *******
    // recurrent weights Wr:
    // 1) [nOut, 4*nOut]    when directionMode <  2
    // 2) [2, nOut, 4*nOut] when directionMode >= 2

    // *******
    // peephole weights Wp:
    // 1) [3*nOut]    when directionMode <  2
    // 2) [2, 3*nOut] when directionMode >= 2

    // *******
    // biases b:
    // 1) [4*nOut]    when directionMode <  2
    // 2) [2, 4*nOut] when directionMode >= 2

    // *******
    // sequence length array seqLen:
    // 1) [bS] always

    // *******
    // initial output hI:
    // 1) [bS, nOut]    when directionMode <  2
    // 2) [2, bS, nOut] when directionMode >= 2

    // *******
    // initial cell state cI (same shape as in hI):
    // 1) [bS, nOut]    when directionMode <  2
    // 2) [2, bS, nOut] when directionMode >= 2


    //     OUTPUTS:

    // *******
    // output h:
    // 1) [sL, bS, nOut]    when directionMode <= 2 && dataFormat == 0
    // 2) [bS, sL, nOut]    when directionMode <= 2 && dataFormat == 1
    // 3) [bS, nOut, sL]    when directionMode <= 2 && dataFormat == 2
    // 4) [sL, bS, 2*nOut]  when directionMode == 3 && dataFormat == 0
    // 5) [bS, sL, 2*nOut]  when directionMode == 3 && dataFormat == 1
    // 6) [bS, 2*nOut, sL]  when directionMode == 3 && dataFormat == 2
    // 7) [sL, 2, bS, nOut] when directionMode == 4 && dataFormat == 3

    // *******
    // output at last step hL:
    // 1) [bS, nOut]    when directionMode <  2
    // 2) [2, bS, nOut] when directionMode >= 2

    // *******
    // cell state at last step cL (same shape as in hL):
    // 1) [bS, nOut]    when directionMode <  2
    // 2) [2, bS, nOut] when directionMode >= 2

    // !!! dimension 4*nOut implies order it, ft, c't, ot
    // !!! dimension 3*nOut implies order it, ft, ot

    const auto dataFormat    = INT_ARG(0);    // for unidirectional: 0 = [sL, bS, nIn], 1 = [bS, sL ,nIn], 2 = [bS, nIn, sL], for bidirectional: 3 = [sL, 2, bS, nOut] (ONNX)
    const auto directionMode = INT_ARG(1);    // direction: 0 = fwd, 1 = bwd, 2 = bidirectional sum, 3 = bidirectional concat, 4 = bidirectional extra output dim (in conjunction with format dataFormat = 3)

    // integer numbers corresponding to activations: 0=tanh, 1=relu, 2=sigmoid, 3=affine, 4=leaky relu, 5= thresholded relu, 6=scaled tanh, 7=hard sigmoid, 8=ELU, 9=softsign, 10=softplus
    const auto gateAct       = INT_ARG(2);    // activation for input (i), forget (f) and output (o) gates
    const auto cellAct       = INT_ARG(3);    // activation for cell state (c')
    const auto outAct        = INT_ARG(4);    // activation for output (h)

    const auto hasBiases  = B_ARG(0);   // indicates whether biases array is provided
    const auto hasSeqLen  = B_ARG(1);   // indicates whether seqLen array is provided
    const auto hasInitH   = B_ARG(2);   // indicates whether initial output is provided
    const auto hasInitC   = B_ARG(3);   // indicates whether initial cell state is provided
    const auto hasPH      = B_ARG(4);   // indicates whether peephole connections are present
    const auto retFullSeq = B_ARG(5);   // indicates whether to return whole time sequence h {h_0, h_1, ... , h_sL-1}
    const auto retLastH   = B_ARG(6);   // indicates whether to return output at last time step only, in this case shape would be [bS, nOut] (exact shape depends on dataFormat argument)
    const auto retLastC   = B_ARG(7);   // indicates whether to return cells state at last time step only, in this case shape would be [bS, nOut] (exact shape depends on dataFormat argument)

    const auto gateActHasAlpha = gateAct == 3 || gateAct == 4 || gateAct == 5 || gateAct == 6 || gateAct == 8;
    const auto cellActHasAlpha = cellAct == 3 || cellAct == 4 || cellAct == 5 || cellAct == 6 || cellAct == 8;
    const auto outActHasAlpha  = outAct  == 3 || outAct  == 4 || outAct  == 5 || outAct  == 6 || outAct  == 8;
    const auto gateActHasBeta  = gateAct == 3 || gateAct == 6;
    const auto cellActHasBeta  = cellAct == 3 || cellAct == 6;
    const auto outActHasBeta   = outAct  == 3 || outAct  == 6;

    uint count = 1;
    const auto cellClip = T_ARG(0);                                     // cell clipping value, if it = 0 then do not apply clipping
    const auto gateAlpha = gateActHasAlpha ? T_ARG(count++) : 0;
    const auto gateBeta  = gateActHasBeta  ? T_ARG(count++) : 0;
    const auto cellAlpha = cellActHasAlpha ? T_ARG(count++) : 0;
    const auto cellBeta  = cellActHasBeta  ? T_ARG(count++) : 0;
    const auto outAlpha  = outActHasAlpha  ? T_ARG(count++) : 0;
    const auto outBeta   = outActHasBeta   ? T_ARG(count++) : 0;

    const auto x  = INPUT_VARIABLE(0);          // input
    const auto Wx = INPUT_VARIABLE(1);          // input weights
    const auto Wr = INPUT_VARIABLE(2);          // recurrent weights

    count = 3;
    const auto b      = hasBiases ? INPUT_VARIABLE(count++) : nullptr;  // biases
    const auto seqLen = hasSeqLen ? INPUT_VARIABLE(count++) : nullptr;  // seqLen vector
    const auto hI     = hasInitH  ? INPUT_VARIABLE(count++) : nullptr;  // initial output
    const auto cI     = hasInitC  ? INPUT_VARIABLE(count++) : nullptr;  // initial cell state
    const auto Wp     = hasPH     ? INPUT_VARIABLE(count++) : nullptr;  // peephole weights

    REQUIRE_TRUE(dataFormat < 3 || (dataFormat == 3 && directionMode == 4), 0, "LSTM_LAYER CUDNN operation: if argument dataFormat = 3, then directionMode = 4, but got dataFormat = %i and directionMode = %i instead !", dataFormat, directionMode);
    REQUIRE_TRUE(cellClip >= 0 , 0, "LSTM_LAYER CUDNN operation: cell clipping value should be nonnegative (>=0) !");
    REQUIRE_TRUE(retFullSeq || retLastH || retLastC, 0, "LSTM_LAYER CUDNN operation: please specify what output arrays to produce !");

    count = 0;
    auto h  = retFullSeq ? OUTPUT_VARIABLE(count++) : nullptr;           // output
    auto hL = retLastH   ? OUTPUT_VARIABLE(count++) : nullptr;           // output at last step
    auto cL = retLastC   ? OUTPUT_VARIABLE(count++) : nullptr;           // cell state at last step

    // evaluate dimensions
    const Nd4jLong sL   = dataFormat == 3 ?  x->sizeAt(0) : x->sizeAt(dataFormat);
    const Nd4jLong bS   = dataFormat == 1 || dataFormat == 2 ? x->sizeAt(0) : x->sizeAt(-2);
    const Nd4jLong nIn  = dataFormat == 2 ? x->sizeAt(1) : x->sizeAt(-1);
    const Nd4jLong nOut = Wx->sizeAt(-1) / 4;

    // inputs validations
    if(directionMode < 2) {     // no bidirectional

        // Wx validation
        if(Wx->rankOf() != 2 || Wx->sizeAt(0) != nIn)
            REQUIRE_TRUE(false, 0, "LSTM_LAYER CUDNN operation: wrong shape of input weights, expected is %s, but got %s instead !", ShapeUtils::shapeAsString({nIn, 4*nOut}).c_str(), ShapeUtils::shapeAsString(Wx).c_str());
        // Wr validation
        if(Wr->rankOf() != 2 || Wr->sizeAt(0) != nOut || Wr->sizeAt(1) != 4*nOut)
            REQUIRE_TRUE(false, 0, "LSTM_LAYER CUDNN operation: wrong shape of recurrent weights, expected is %s, but got %s instead !", ShapeUtils::shapeAsString({nOut, 4*nOut}).c_str(), ShapeUtils::shapeAsString(Wr).c_str());
        // biases validation
        if(b != nullptr && (b->rankOf() != 1 || b->sizeAt(0) != 4*nOut))
            REQUIRE_TRUE(false, 0, "LSTM_LAYER CUDNN operation: wrong shape of biases, expected is %s, but got %s instead !", ShapeUtils::shapeAsString({4*nOut}).c_str(), ShapeUtils::shapeAsString(b).c_str());
        // initial output validation
        if(hI != nullptr && (hI->rankOf() != 2 || hI->sizeAt(0) != bS || hI->sizeAt(1) != nOut))
            REQUIRE_TRUE(false, 0, "LSTM_LAYER CUDNN operation: wrong shape of initial output, expected is %s, but got %s instead !", ShapeUtils::shapeAsString({bS, nOut}).c_str(), ShapeUtils::shapeAsString(hI).c_str());
        // initial cell  validation
        if(cI != nullptr && (cI->rankOf() != 2 || cI->sizeAt(0) != bS || cI->sizeAt(1) != nOut))
            REQUIRE_TRUE(false, 0, "LSTM_LAYER CUDNN operation: wrong shape of initial cell state, expected is %s, but got %s instead !", ShapeUtils::shapeAsString({bS, nOut}).c_str(), ShapeUtils::shapeAsString(cI).c_str());
    }
    else {                  // bidirectional
         // Wx validation
        if(Wx->rankOf() != 3 || Wx->sizeAt(0) != 2 || Wx->sizeAt(1) != nIn)
            REQUIRE_TRUE(false, 0, "LSTM_LAYER CUDNN operation: wrong shape of input weights, expected is %s, but got %s instead !", ShapeUtils::shapeAsString({2, nIn, 4*nOut}).c_str(), ShapeUtils::shapeAsString(Wx).c_str());
        // Wr validation
        if(Wr->rankOf() != 3 || Wr->sizeAt(0) != 2 || Wr->sizeAt(1) != nOut || Wr->sizeAt(2) != 4*nOut)
            REQUIRE_TRUE(false, 0, "LSTM_LAYER CUDNN operation: wrong shape of recurrent weights, expected is %s, but got %s instead !", ShapeUtils::shapeAsString({2, nOut, 4*nOut}).c_str(), ShapeUtils::shapeAsString(Wr).c_str());
        // biases validation
        if(b != nullptr && (b->rankOf() != 2 || b->sizeAt(0) != 2 || b->sizeAt(1) != 4*nOut))
            REQUIRE_TRUE(false, 0, "LSTM_LAYER CUDNN operation: wrong shape of biases, expected is %s, but got %s instead !", ShapeUtils::shapeAsString({2, 4*nOut}).c_str(), ShapeUtils::shapeAsString(b).c_str());
        // initial output validation
        if(hI != nullptr && (hI->rankOf() != 3 || hI->sizeAt(0) != 2 || hI->sizeAt(1) != bS || hI->sizeAt(2) != nOut))
            REQUIRE_TRUE(false, 0, "LSTM_LAYER CUDNN operation: wrong shape of initial output, expected is %s, but got %s instead !", ShapeUtils::shapeAsString({2, bS, nOut}).c_str(), ShapeUtils::shapeAsString(hI).c_str());
        // initial cell  validation
        if(cI != nullptr && (cI->rankOf() != 3 || cI->sizeAt(0) != 2 || cI->sizeAt(1) != bS || cI->sizeAt(2) != nOut))
            REQUIRE_TRUE(false, 0, "LSTM_LAYER CUDNN operation: wrong shape of initial cell state, expected is %s, but got %s instead !", ShapeUtils::shapeAsString({2, bS, nOut}).c_str(), ShapeUtils::shapeAsString(cI).c_str());
    }

    std::vector<float> params = {static_cast<float>(dataFormat), static_cast<float>(directionMode), static_cast<float>(cellClip),
                                 static_cast<float>(gateAct), static_cast<float>(gateAlpha), static_cast<float>(gateBeta),
                                 static_cast<float>(cellAct), static_cast<float>(cellAlpha), static_cast<float>(cellBeta),
                                 static_cast<float>(outAct), static_cast<float>(outAlpha), static_cast<float>(outBeta)};

    const uint numOfDirections = (directionMode < 2) ? 1 : 2;

    // cudnn requires only one weights array
    NDArray w(Wx->ordering(), {numOfDirections * (nIn + nOut), 4*nOut}, Wx->dataType(), Wx->getContext());
    if(numOfDirections == 1)
        helpers::concat(block.launchContext(), {Wx, Wr}, w, 0);
    else {
        NDArray WxForward  = (*Wx)({0,1, 0,0, 0,0});
        NDArray WrForward  = (*Wr)({0,1, 0,0, 0,0});
        NDArray WxBackward = (*Wx)({1,2, 0,0, 0,0});
        NDArray WrBackward = (*Wr)({1,2, 0,0, 0,0});

        helpers::concat(block.launchContext(), {&WxForward, &WrForward, &WxBackward, &WrBackward}, w, 0);  // nIn + nOut + nIn + nOut = 2 * (nIn + nOut)
        // w({0,nIn,    0,0}).assign((*Wx)({0,1, 0,0, 0,0}));
        // w({nIn,nOut, 0,0}).assign((*Wr)({0,1, 0,0, 0,0}));
        // w({nIn+nOut,2*nIn+nOut, 0,0}).assign((*Wx)({1,2, 0,0, 0,0}));
        // w({2*nIn+nOut,-1,       0,0}).assign((*Wr)({1,2, 0,0, 0,0}));
    }

    lstmLayerCUDNN(block.launchContext(), x, &w, b, seqLen, hI, cI, h, hL, cL, params);

    return Status::OK();
}

//////////////////////////////////////////////////////////////////////////
PLATFORM_CHECK(lstmLayer, ENGINE_CUDA) {

    const auto x = INPUT_VARIABLE(0);
    const auto Wx = INPUT_VARIABLE(1);
    const auto Wr = INPUT_VARIABLE(2);

    const auto dataFormat    = INT_ARG(0);    // for unidirectional: 0 = [sL, bS, nIn], 1 = [bS, sL ,nIn], 2 = [bS, nIn, sL], for bidirectional: 3 = [sL, 2, bS, nOut] (ONNX)
    const auto directionMode = INT_ARG(1);    // direction: 0 = fwd, 1 = bwd, 2 = bidirectional sum, 3 = bidirectional concat, 4 = bidirectional extra output dim (in conjunction with format dataFormat = 3)

    if(dataFormat != 2 || (directionMode != 0 && directionMode != 3))
        return false;

    const auto gateAct = INT_ARG(2);    // activation for input (i), forget (f) and output (o) gates, 2==sigmoid is supported only
    const auto cellAct = INT_ARG(3);    // activation for cell state (c'), 0==tanh is supported only
    const auto outAct  = INT_ARG(4);    // activation for output (h), 0==tanh is supported only

    if(gateAct != 2 || cellAct != 0 || outAct != 0)
        return false;

    const auto hasBiases  = B_ARG(0);   // indicates whether biases array is provided
    const auto hasSeqLen  = B_ARG(1);   // indicates whether seqLen array is provided
    const auto hasInitH   = B_ARG(2);   // indicates whether initial output is provided
    const auto hasInitC   = B_ARG(3);   // indicates whether initial cell state is provided
    const auto hasPH      = B_ARG(4);   // indicates whether peephole connections are present
    const auto retFullSeq = B_ARG(5);   // indicates whether to return whole time sequence h {h_0, h_1, ... , h_sL-1}
    const auto retLastH   = B_ARG(6);   // indicates whether to return output at last time step only, in this case shape would be [bS, nOut] (exact shape depends on dataFormat argument)
    const auto retLastC   = B_ARG(7);   // indicates whether to return cells state at last time step only, in this case shape would be [bS, nOut] (exact shape depends on dataFormat argument)

    if(hasPH || !retFullSeq)
        return false;

    uint count = 3;
    const auto b      = hasBiases ? INPUT_VARIABLE(count++) : nullptr;  // biases
    const auto seqLen = hasSeqLen ? INPUT_VARIABLE(count++) : nullptr;  // seqLen vector
    const auto hI     = hasInitH  ? INPUT_VARIABLE(count++) : nullptr;  // initial output
    const auto cI     = hasInitC  ? INPUT_VARIABLE(count++) : nullptr;  // initial cell state
    const auto Wp     = hasPH     ? INPUT_VARIABLE(count++) : nullptr;  // peephole weights

    count = 0;
    auto h  = retFullSeq ? OUTPUT_VARIABLE(count++) : nullptr;           // output
    auto hL = retLastH   ? OUTPUT_VARIABLE(count++) : nullptr;           // output at last step
    auto cL = retLastC   ? OUTPUT_VARIABLE(count++) : nullptr;           // cell state at last step

    if(x->ews() != 1)
        return false;

    if(hasInitH && hI->ews() != 1)
        return false;

    if(hasInitC && cI->ews() != 1)
        return false;

    if(h->ews() != 1)
        return false;

    if(retLastH && hL->ews() != 1)
        return false;

    if(retLastC && cL->ews() != 1)
        return false;

    const auto xDataType = x->dataType();

    bool goodType = xDataType == DataType::DOUBLE || xDataType == DataType::FLOAT32 || xDataType != DataType::HALF;
    goodType &= Wx->dataType() == xDataType && Wr->dataType() == xDataType;
    if(hasBiases)
        goodType &= b->dataType() == xDataType;
    if(hasSeqLen)
        goodType &= seqLen->dataType() == DataType::INT32;
    if(hasInitH)
        goodType &= hI->dataType() == xDataType;
    if(hasInitC)
        goodType &= cI->dataType() == xDataType;

    if(!goodType)
        return false;

    return true;
}



}
}
}
