#include "hip/hip_runtime.h"
/*******************************************************************************
 * Copyright (c) 2019 Konduit K.K.
 *
 * This program and the accompanying materials are made available under the
 * terms of the Apache License, Version 2.0 which is available at
 * https://www.apache.org/licenses/LICENSE-2.0.
 *
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS, WITHOUT
 * WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied. See the
 * License for the specific language governing permissions and limitations
 * under the License.
 *
 * SPDX-License-Identifier: Apache-2.0
 ******************************************************************************/

//
// @author Yurii Shyrma (iuriish@yahoo.com)
//


#include "cudnnUtils.h"
#include <ops/declarable/helpers/convolutions.h>

namespace nd4j      {
namespace ops       {
namespace platforms {

//////////////////////////////////////////////////////////////////////////
static void lstmLayerCUDNN() {


}

//////////////////////////////////////////////////////////////////////////
PLATFORM_IMPL(lstmLayer, ENGINE_CUDA) {

     // it  = σ(Wxi * xt  +  Wri * ht-1  +  bi)
    // ft  = σ(Wxf * xt  +  Wrf * ht-1  +  bf)
    // c't = tanh(Wxc * xt  +  Wrc * ht-1  +  bc)
    // ct  = ft ◦ ct-1 + it ◦ c't
    // ot  = σ(Wxo * xt  +  Wro * ht-1  +  bo)
    // ht  = ot ◦ tanh(ct)

    // equations (peephole connections are present)
    // it  = σ(Wxi * xt  +  Wri * ht-1  +  Wpi ◦ ct-1  +  bi)
    // ft  = σ(Wxf * xt  +  Wrf * ht-1  +  Wpf ◦ ct-1  +  bf)
    // c't = tanh(Wxc * xt  +  Wrc * ht-1  +  bc)
    // ct  = ft ◦ ct-1 + it ◦ c't
    // ot  = σ(Wxo * xt  +  Wro * ht-1  +  Wpo ◦ ct  +  bo)
    // ht  = ot ◦ tanh(ct)

    // notations:
    // bS - batch size
    // sL - sequence length, number of time steps
    // nIn - input size
    // nOut - output size (hidden size)

    //     INPUTS:

    // *******
    // input x:
    // 1) [sL, bS, nIn]  when dataFormat == 0
    // 2) [bS, sL, nIn]  when dataFormat == 1
    // 3) [bS, nIn, sL]  when dataFormat == 2

    // *******
    // input weights Wx:
    // 1) [nIn, 4*nOut]    when directionMode <  2
    // 2) [2, nIn, 4*nOut] when directionMode >= 2

    // *******
    // recurrent weights Wr:
    // 1) [nOut, 4*nOut]    when directionMode <  2
    // 2) [2, nOut, 4*nOut] when directionMode >= 2

    // *******
    // peephole weights Wp:
    // 1) [3*nOut]    when directionMode <  2
    // 2) [2, 3*nOut] when directionMode >= 2

    // *******
    // biases b:
    // 1) [4*nOut]    when directionMode <  2
    // 2) [2, 4*nOut] when directionMode >= 2

    // *******
    // sequence length array seqLen:
    // 1) [bS] always

    // *******
    // initial output hI:
    // 1) [bS, nOut]    when directionMode <  2
    // 2) [2, bS, nOut] when directionMode >= 2

    // *******
    // initial cell state cI (same shape as in hI):
    // 1) [bS, nOut]    when directionMode <  2
    // 2) [2, bS, nOut] when directionMode >= 2


    //     OUTPUTS:

    // *******
    // output h:
    // 1) [sL, bS, nOut]    when directionMode <= 2 && dataFormat == 0
    // 2) [bS, sL, nOut]    when directionMode <= 2 && dataFormat == 1
    // 3) [bS, nOut, sL]    when directionMode <= 2 && dataFormat == 2
    // 4) [sL, bS, 2*nOut]  when directionMode == 3 && dataFormat == 0
    // 5) [bS, sL, 2*nOut]  when directionMode == 3 && dataFormat == 1
    // 6) [bS, 2*nOut, sL]  when directionMode == 3 && dataFormat == 2
    // 7) [sL, 2, bS, nOut] when directionMode == 4 && dataFormat == 3

    // *******
    // output at last step hL:
    // 1) [bS, nOut]    when directionMode <  2
    // 2) [2, bS, nOut] when directionMode >= 2

    // *******
    // cell state at last step cL (same shape as in hL):
    // 1) [bS, nOut]    when directionMode <  2
    // 2) [2, bS, nOut] when directionMode >= 2

    // !!! dimension 4*nOut implies order it, ft, c't, ot
    // !!! dimension 3*nOut implies order it, ft, ot

}

//////////////////////////////////////////////////////////////////////////
PLATFORM_CHECK(lstmLayer, ENGINE_CUDA) {

    const auto x = INPUT_VARIABLE(0);
    const auto Wx = INPUT_VARIABLE(1);
    const auto Wr = INPUT_VARIABLE(2);

    const auto dataFormat    = INT_ARG(0);    // for unidirectional: 0 = [sL, bS, nIn], 1 = [bS, sL ,nIn], 2 = [bS, nIn, sL], for bidirectional: 3 = [sL, 2, bS, nOut] (ONNX)
    const auto directionMode = INT_ARG(1);    // direction: 0 = fwd, 1 = bwd, 2 = bidirectional sum, 3 = bidirectional concat, 4 = bidirectional extra output dim (in conjunction with format dataFormat = 3)

    const auto hasBiases  = B_ARG(0);   // indicates whether biases array is provided
    const auto hasSeqLen  = B_ARG(1);   // indicates whether seqLen array is provided
    const auto hasInitH   = B_ARG(2);   // indicates whether initial output is provided
    const auto hasInitC   = B_ARG(3);   // indicates whether initial cell state is provided
    const auto hasPH      = B_ARG(4);   // indicates whether peephole connections are present
    const auto retFullSeq = B_ARG(5);   // indicates whether to return whole time sequence h {h_0, h_1, ... , h_sL-1}
    const auto retLastH   = B_ARG(6);   // indicates whether to return output at last time step only, in this case shape would be [bS, nOut] (exact shape depends on dataFormat argument)
    const auto retLastC   = B_ARG(7);   // indicates whether to return cells state at last time step only, in this case shape would be [bS, nOut] (exact shape depends on dataFormat argument)

    if(directionMode == 2 || directionMode == 4)
        return false;

    if(hasPH)
        return false;

    uint count = 3;
    const auto b      = hasBiases ? INPUT_VARIABLE(count++) : nullptr;  // biases
    const auto seqLen = hasSeqLen ? INPUT_VARIABLE(count++) : nullptr;  // seqLen vector
    const auto hI     = hasInitH  ? INPUT_VARIABLE(count++) : nullptr;  // initial output
    const auto cI     = hasInitC  ? INPUT_VARIABLE(count++) : nullptr;  // initial cell state
    const auto Wp     = hasPH     ? INPUT_VARIABLE(count++) : nullptr;  // peephole weights

    count = 0;
    auto h  = retFullSeq ? OUTPUT_VARIABLE(count++) : nullptr;           // output
    auto hL = retLastH   ? OUTPUT_VARIABLE(count++) : nullptr;           // output at last step
    auto cL = retLastC   ? OUTPUT_VARIABLE(count++) : nullptr;           // cell state at last step

    if(hasInitH && hI->ews() != 1)
        return false;

    if(hasInitC && cI->ews() != 1)
        return false;

    if(retFullSeq && h->ews() != 1)
        return false;

    if(retLastH && hL->ews() != 1)
        return false;

    if(retLastC && cL->ews() != 1)
        return false;

    const auto xDataType = x->dataType();

    bool goodType = xDataType == DataType::DOUBLE || xDataType == DataType::FLOAT32 || xDataType != DataType::HALF;
    goodType &= Wx->dataType() == xDataType && Wr->dataType() == xDataType;
    if(hasBiases)
        goodType &= b->dataType() == xDataType;
    if(hasSeqLen)
        goodType &= seqLen->dataType() == DataType::INT32;
    if(hasInitH)
        goodType &= hI->dataType() == xDataType;
    if(hasInitC)
        goodType &= cI->dataType() == xDataType;

    if(!goodType)
        return false;




}



}
}
}
