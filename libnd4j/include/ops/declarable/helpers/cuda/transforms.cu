#include "hip/hip_runtime.h"
/*******************************************************************************
 * Copyright (c) 2015-2018 Skymind, Inc.
 * Copyright (c) 2019 Konduit K.K.
 *
 * This program and the accompanying materials are made available under the
 * terms of the Apache License, Version 2.0 which is available at
 * https://www.apache.org/licenses/LICENSE-2.0.
 *
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS, WITHOUT
 * WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied. See the
 * License for the specific language governing permissions and limitations
 * under the License.
 *
 * SPDX-License-Identifier: Apache-2.0
 ******************************************************************************/

//
// @author Yurii Shyrma (iuriish@yahoo.com), created on 20.04.2018
//

#include <array/NDArrayFactory.h>
#include <array/ResultSet.h>
#include <exceptions/cuda_exception.h>
#include <helpers/ConstantTadHelper.h>
#include <helpers/PointersManager.h>
#include <helpers/ShapeUtils.h>
#include <helpers/TAD.h>
#include <ops/declarable/helpers/transforms.h>

#include <numeric>

namespace sd {
namespace ops {
namespace helpers {

///////////////////////////////////////////////////////////////////
template <typename T>
__global__ static void invertPermutationCuda(const void* vx,
                                             const Nd4jLong* xShapeInfo,
                                             void* vz,
                                             const Nd4jLong* zShapeInfo) {
  const T* x = reinterpret_cast<const T*>(vx);
  T* z = reinterpret_cast<T*>(vz);

  __shared__ Nd4jLong len, totalThreads;

  if (threadIdx.x == 0) {
    len = shape::length(xShapeInfo);
    totalThreads = gridDim.x * blockDim.x;
  }

  __syncthreads();

  const auto tid = blockIdx.x * blockDim.x + threadIdx.x;

  for (Nd4jLong i = tid; i < len; i += totalThreads) {
    const auto xOffset = shape::getIndexOffset(i, xShapeInfo);
    const Nd4jLong index = x[xOffset];
    const auto zOffset = shape::getIndexOffset(index, zShapeInfo);
    z[zOffset] = i;
  }
}

///////////////////////////////////////////////////////////////////
template <typename T>
__host__ static void invertPermutationCudaLauncher(
    const int blocksPerGrid, const int threadsPerBlock,
    const hipStream_t* stream, const void* vx, const Nd4jLong* xShapeInfo,
    void* vz, const Nd4jLong* zShapeInfo) {
  invertPermutationCuda<T><<<blocksPerGrid, threadsPerBlock, 1024, *stream>>>(
      vx, xShapeInfo, vz, zShapeInfo);
}

////////////////////////////////////////////////////////////////////////
void invertPermutation(sd::LaunchContext* context, const NDArray& input,
                       NDArray& output) {
  const int threadsPerBlock = MAX_NUM_THREADS;
  const int blocksPerGrid =
      (input.lengthOf() + threadsPerBlock - 1) / threadsPerBlock;

  PointersManager manager(context, "invertPermutation");

  NDArray::prepareSpecialUse({&output}, {&input});
  BUILD_SINGLE_SELECTOR(
      input.dataType(), invertPermutationCudaLauncher,
      (blocksPerGrid, threadsPerBlock, context->getCudaStream(),
       input.specialBuffer(), input.specialShapeInfo(), output.specialBuffer(),
       output.specialShapeInfo()),
      LIBND4J_TYPES);
  NDArray::registerSpecialUse({&output}, {&input});

  manager.synchronize();
}

//////////////////////////////////////////////////////////////////////////
template <typename T>
__global__ static void traceCuda(const void* vx, const Nd4jLong* xShapeInfo,
                                 void* vz, const Nd4jLong* zShapeInfo,
                                 const uint diagLen) {
  const auto x = reinterpret_cast<const T*>(vx);
  auto z = reinterpret_cast<T*>(vz);

  __shared__ T* sharedMem;
  __shared__ int xRank, zRank, *coordsMem;  // xRank = zRank + 2
  __shared__ Nd4jLong xLen, zLen;

  if (threadIdx.x == 0) {
    extern __shared__ unsigned char shmem[];
    sharedMem = reinterpret_cast<T*>(shmem);
    coordsMem = reinterpret_cast<int*>(shmem + blockDim.x * sizeof(T));

    xRank = shape::rank(xShapeInfo);
    zRank = shape::rank(zShapeInfo);
    xLen = shape::length(xShapeInfo);
    zLen = shape::length(zShapeInfo);  // corresponds to number of matrices
  }
  __syncthreads();

  auto coords = coordsMem + threadIdx.x * xRank;

  for (uint m = blockIdx.x; m < zLen;
       m +=
       gridDim.x) {  // one block per each element of z, that is per each matrix

    shape::index2coords(m, zShapeInfo, coords);
    const auto zOffset = shape::getOffset(zShapeInfo, coords);

    sharedMem[threadIdx.x] = 0;

    for (uint i = threadIdx.x; i < diagLen; i += blockDim.x) {
      coords[zRank] = coords[zRank + 1] = i;
      const auto xOffset = shape::getOffset(xShapeInfo, coords);
      sharedMem[threadIdx.x] += x[xOffset];
    }

    __syncthreads();

    // aggregate sum
    for (Nd4jLong activeThreads = blockDim.x / 2; activeThreads > 0;
         activeThreads /= 2) {
      if (threadIdx.x < activeThreads)
        sharedMem[threadIdx.x] += sharedMem[threadIdx.x + activeThreads];
      __syncthreads();
    }

    if (threadIdx.x == 0) z[zOffset] = *sharedMem;
    __syncthreads();
  }
}

///////////////////////////////////////////////////////////////////
template <typename T>
static void traceCudaLauncher(const int blocksPerGrid,
                              const int threadsPerBlock, const int sharedMem,
                              const hipStream_t* stream, const void* vx,
                              const Nd4jLong* xShapeInfo, void* vz,
                              const Nd4jLong* zShapeInfo, const uint diagLen) {
  traceCuda<T><<<blocksPerGrid, threadsPerBlock, sharedMem, *stream>>>(
      vx, xShapeInfo, vz, zShapeInfo, diagLen);
}

///////////////////////////////////////////////////////////////////
void trace(sd::LaunchContext* context, const NDArray& input, NDArray& output) {
  PointersManager manager(context, "trace");

  const uint diagLen =
      input.sizeAt(-1) < input.sizeAt(-2) ? input.sizeAt(-1) : input.sizeAt(-2);
  const int threadsPerBlock = MAX_NUM_THREADS / 4;
  const int blocksPerGrid =
      (output.lengthOf() + threadsPerBlock - 1) / threadsPerBlock;
  const int sharedMem =
      threadsPerBlock * (sizeof(int) * input.rankOf() + input.sizeOfT()) + 128;

  NDArray::prepareSpecialUse({&output}, {&input});
  BUILD_SINGLE_SELECTOR(
      input.dataType(), traceCudaLauncher,
      (blocksPerGrid, threadsPerBlock, sharedMem, context->getCudaStream(),
       input.specialBuffer(), input.specialShapeInfo(), output.specialBuffer(),
       output.specialShapeInfo(), diagLen),
      LIBND4J_TYPES);
  NDArray::registerSpecialUse({&output}, {&input});

  manager.synchronize();
}

///////////////////////////////////////////////////////////////////
template <typename T>
__global__ static void triuBPCuda(const void* vx, const Nd4jLong* xShapeInfo,
                                  void* vz, const Nd4jLong* zShapeInfo,
                                  const int diag) {
  // x and z have same shapes
  const auto x = reinterpret_cast<const T*>(vx);  // gradO
  auto z = reinterpret_cast<T*>(vz);              // gradI

  __shared__ int rank, areSameOffsets, *sharedMem;  // xRank = zRank
  __shared__ Nd4jLong len, totalThreads;            // xLen = zLen

  if (threadIdx.x == 0) {
    extern __shared__ unsigned char shmem[];
    sharedMem = reinterpret_cast<int*>(shmem);
    areSameOffsets = shape::haveSameShapeAndStrides(xShapeInfo, zShapeInfo);
    rank = shape::rank(xShapeInfo);
    len = shape::length(zShapeInfo);
    totalThreads = gridDim.x * blockDim.x;
  }

  __syncthreads();

  auto coords = sharedMem + threadIdx.x * rank;

  const auto tid = blockIdx.x * blockDim.x + threadIdx.x;

  for (Nd4jLong i = tid; i < len; i += totalThreads) {
    shape::index2coords(i, zShapeInfo, coords);

    const auto zOffset = shape::getOffset(zShapeInfo, coords);

    if ((coords[rank - 2] + diag > coords[rank - 1]))  // row + diag > col
      z[zOffset] = 0;
    else
      z[zOffset] =
          x[areSameOffsets ? zOffset : shape::getOffset(xShapeInfo, coords)];
  }
}

///////////////////////////////////////////////////////////////////
template <typename T>
static void triuBPCudaLauncher(const int blocksPerGrid,
                               const int threadsPerBlock, const int sharedMem,
                               const hipStream_t* stream, const void* vx,
                               const Nd4jLong* xShapeInfo, void* vz,
                               const Nd4jLong* zShapeInfo, const int diag) {
  triuBPCuda<T><<<blocksPerGrid, threadsPerBlock, sharedMem, *stream>>>(
      vx, xShapeInfo, vz, zShapeInfo, diag);
}

///////////////////////////////////////////////////////////////////
void triuBP(sd::LaunchContext* context, const NDArray& input,
            const NDArray& gradO, NDArray& gradI, const int diagonal) {
  const int threadsPerBlock = MAX_NUM_THREADS / 4;
  const int blocksPerGrid =
      (gradO.lengthOf() + threadsPerBlock - 1) / threadsPerBlock;
  const int sharedMem = threadsPerBlock * sizeof(int) * gradO.rankOf() + 128;

  PointersManager manager(context, "triuBP");

  NDArray::prepareSpecialUse({&gradI}, {&gradO});
  BUILD_SINGLE_SELECTOR(
      gradI.dataType(), triuBPCudaLauncher,
      (blocksPerGrid, threadsPerBlock, sharedMem, context->getCudaStream(),
       gradO.specialBuffer(), gradO.specialShapeInfo(), gradI.specialBuffer(),
       gradI.specialShapeInfo(), diagonal),
      LIBND4J_TYPES);
  NDArray::registerSpecialUse({&gradI}, {&gradO});

  manager.synchronize();
}

///////////////////////////////////////////////////////////////////
template <typename T>
__global__ static void tileBPCuda(const void* vx, const Nd4jLong* xShapeInfo,
                                  void* vz, const Nd4jLong* zShapeInfo,
                                  Nd4jLong* globMem) {
  // x and z have same shapes
  const auto x = reinterpret_cast<const T*>(vx);  // gradO
  auto z = reinterpret_cast<T*>(vz);              // gradI

  __shared__ int xRank, zRank, *sharedMem;                // xRank >= zRank
  __shared__ Nd4jLong numOfXOffsets, zLen, totalThreads;  // xLen >= zLen

  if (threadIdx.x == 0) {
    extern __shared__ unsigned char shmem[];
    sharedMem = reinterpret_cast<int*>(shmem);

    xRank = shape::rank(zShapeInfo);
    zLen = shape::length(zShapeInfo);
    numOfXOffsets = shape::length(xShapeInfo) / zLen;

    totalThreads = gridDim.x * blockDim.x;
  }

  __syncthreads();

  const auto tid = blockIdx.x * blockDim.x + threadIdx.x;

  auto memBuff = sharedMem + threadIdx.x * 2 * xRank;
  auto xOffsets = globMem + tid * numOfXOffsets;

  for (Nd4jLong i = tid; i < zLen; i += totalThreads) {
    const auto zOffset = shape::getIndexOffset(i, zShapeInfo);

    shape::outerArrayOffsets(xOffsets, i, xShapeInfo, zShapeInfo, memBuff);

    z[zOffset] = x[xOffsets[0]];                  // first offset
    for (Nd4jLong j = 1; j < numOfXOffsets; ++j)  // rest offsets
      z[zOffset] += x[xOffsets[j]];
  }
}

///////////////////////////////////////////////////////////////////
template <typename T>
static void tileBPCudaLauncher(const int blocksPerGrid,
                               const int threadsPerBlock, const int sharedMem,
                               const hipStream_t* stream, const void* vx,
                               const Nd4jLong* xShapeInfo, void* vz,
                               const Nd4jLong* zShapeInfo, Nd4jLong* globMem) {
  tileBPCuda<T><<<blocksPerGrid, threadsPerBlock, sharedMem, *stream>>>(
      vx, xShapeInfo, vz, zShapeInfo, globMem);
}

//////////////////////////////////////////////////////////////////////////
void tileBP(sd::LaunchContext* context, const NDArray& gradO /*input*/,
            NDArray& gradI /*output*/, const std::vector<Nd4jLong> reps) {
  NDArray memBuff('c', gradO.getShapeAsVector(), sd::DataType::INT64,
                  context);  // empty auxiliary array for storing device memory
                             // which will be used in kernel calculations

  const int threadsPerBlock = MAX_NUM_THREADS / 4;
  const int blocksPerGrid =
      (gradI.lengthOf() + threadsPerBlock - 1) / threadsPerBlock;
  const int sharedMem =
      threadsPerBlock * sizeof(int) * 2 * gradO.rankOf() + 128;

  PointersManager manager(context, "tileBP");

  NDArray::prepareSpecialUse({&gradI}, {&gradO, &memBuff});
  BUILD_SINGLE_SELECTOR(
      gradI.dataType(), tileBPCudaLauncher,
      (blocksPerGrid, threadsPerBlock, sharedMem, context->getCudaStream(),
       gradO.specialBuffer(), gradO.specialShapeInfo(), gradI.specialBuffer(),
       gradI.specialShapeInfo(),
       reinterpret_cast<Nd4jLong*>(memBuff.specialBuffer())),
      FLOAT_TYPES);
  NDArray::registerSpecialUse({&gradI}, {&gradO, &memBuff});

  manager.synchronize();
}

    template <typename T>
    static __global__ void swapShuffleKernel(T* input, Nd4jLong const* shape, Nd4jLong firstDim, sd::graph::RandomGenerator* rng) {
        auto tid = blockIdx.x * blockDim.x;
        auto step = blockDim.x * gridDim.x;

  for (int i = firstDim - 1 - tid - threadIdx.x; i > 0; i -= step) {
    int r = rng->relativeInt(i) % i;
    if (i != r) {
      const auto iOffset = shape::getIndexOffset(i, shape);
      const auto rOffset = shape::getIndexOffset(r, shape);
      T e0 = input[iOffset];
      T e1 = input[rOffset];
      // math::nd4j_swap<T>(input(i), input(r));
      input[iOffset] = e1;
      input[rOffset] = e0;
    }
  }
}
template <typename T>
static __global__ void fillShuffleKernel(T* input, Nd4jLong const* inputShape,
                                         T* output, Nd4jLong const* outputShape,
                                         Nd4jLong firstDim, int* indices,
                                         sd::graph::RandomGenerator* rng) {
  //        PRAGMA_OMP_PARALLEL_FOR_IF((firstDim-1) >
  //        Environment::getInstance().tadThreshold())
  auto tid = blockIdx.x * blockDim.x;
  auto step = blockDim.x * gridDim.x;

  for (int i = firstDim - 1 - tid - threadIdx.x; i > 0; i -= step) {
    int r = rng->relativeInt(i) % i;
    output[shape::getIndexOffset(i, outputShape)] =
        input[shape::getIndexOffset(indices[r], inputShape)];
    if (i != r) {
      output[shape::getIndexOffset(r, outputShape)] =
          input[shape::getIndexOffset(indices[i], inputShape)];
      //                output.p(r, input.e<T>(indices[i]));
      //                math::nd4j_swap<int>(indices[i], indices[r]);
      atomicExch(&indices[i], indices[r]);
    }
  }
}
//////////////////////////////////////////////////////////////////////////
template <typename T>
void randomShuffle_(sd::LaunchContext* context, NDArray& input, NDArray& output,
                    sd::graph::RandomGenerator& rng, const bool isInplace) {
  // check edge cases first
  int temp;
  const int firstDim = input.sizeAt(0);
  auto stream = context->getCudaStream();
  NDArray::prepareSpecialUse({&output}, {&input});
  if (input.lengthOf() == 1 || firstDim == 1) {
    if (!isInplace) output.assign(input);
  } else if (input.isVector() || shape::isLikeVector(input.shapeInfo(), temp)) {
    // apply Fisher-Yates shuffle
    sd::graph::RandomGenerator* dRandom = nullptr;
    hipMalloc(&dRandom, sizeof(sd::graph::RandomGenerator));
    hipMemcpy(dRandom, &rng, sizeof(sd::graph::RandomGenerator),
               hipMemcpyHostToDevice);
    T* inputBuf = reinterpret_cast<T*>(input.specialBuffer());
    if (isInplace) {
      swapShuffleKernel<T><<<128, 256, 1024, *stream>>>(
          inputBuf, input.specialShapeInfo(), firstDim, dRandom);
    } else {
      std::vector<int> indices(firstDim);
      std::iota(indices.begin(), indices.end(), 0);
      hipMemcpy(output.specialBuffer(), input.specialBuffer(), sizeof(T),
                 hipMemcpyDeviceToDevice);
      // output.p<T>(Nd4jLong(0), input.e<T>(0));
      PointersManager pointersManager(context, "helper::randomShuffle_");
      int* indicesDev = reinterpret_cast<int*>(pointersManager.replicatePointer(
          indices.data(), indices.size() * sizeof(int)));
      T* outputBuf = reinterpret_cast<T*>(output.specialBuffer());
      fillShuffleKernel<T><<<128, 256, 1024, *stream>>>(
          inputBuf, input.specialShapeInfo(), outputBuf,
          output.specialShapeInfo(), firstDim, indicesDev, dRandom);
      pointersManager.synchronize();
    }
    //            rng.rewindH(firstDim - 1);
    hipFree(dRandom);
  } else {
    // evaluate sub-arrays list of input array through all dimensions excluding
    // first one
    std::vector<int> dimensions =
        ShapeUtils::evalDimsToExclude(input.rankOf(), {0});
    auto subArrsListIn = input.allTensorsAlongDimension(dimensions);

    // apply Fisher-Yates shuffle
    if (isInplace) {
      for (int i = firstDim - 1; i > 0; --i) {
        int r = rng.relativeInt(i) % i;

        if (i != r) subArrsListIn.at(i)->swapUnsafe(*subArrsListIn.at(r));
      }
    } else {
      // evaluate sub-arrays list of output array through all dimensions
      // excluding first one
      auto subArrsListOut = output.allTensorsAlongDimension(dimensions);
      std::vector<int> indices(firstDim);
      std::iota(indices.begin(), indices.end(), 0);
      bool isZeroShuffled = false;

      for (int i = firstDim - 1; i > 0; --i) {
        int r = rng.relativeInt(i) % i;
        subArrsListOut.at(i)->assign(subArrsListIn.at(indices[r]));
        if (r == 0) isZeroShuffled = true;

        if (i != r) {
          subArrsListOut.at(r)->assign(subArrsListIn.at(indices[i]));
          math::nd4j_swap<int>(indices[i], indices[r]);
        }
      }
      if (!isZeroShuffled) subArrsListOut.at(0)->assign(subArrsListIn.at(0));
    }
    rng.rewindH(firstDim - 1);
  }
  NDArray::registerSpecialUse({&output}, {&input});
}

void randomShuffle(sd::LaunchContext* context, NDArray& input, NDArray& output,
                   sd::graph::RandomGenerator& rng, const bool isInplace) {
  BUILD_SINGLE_SELECTOR(input.dataType(), randomShuffle_,
                        (context, input, output, rng, isInplace),
                        LIBND4J_TYPES);
}

BUILD_SINGLE_TEMPLATE(template void randomShuffle_,
                      (sd::LaunchContext * context, NDArray& input,
                       NDArray& output, sd::graph::RandomGenerator& rng,
                       const bool isInplace),
                      LIBND4J_TYPES);

//////////////////////////////////////////////////////////////////////////
void eye(sd::LaunchContext* context, NDArray& output) { output.setIdentity(); }


}  // namespace helpers
}  // namespace ops
}  // namespace sd
